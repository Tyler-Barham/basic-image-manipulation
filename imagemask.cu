#include <hip/hip_runtime.h>

#include <stdio.h>

extern "C"
int* computeMask( int threshold, int imgArr[], int width, int height );


__global__ void applyMask( int threshold, int length, int *cudaImgArr )
{
    // Determine where in the loop to start
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Determine how quickly to step over the loop
    int stride = blockDim.x * gridDim.x;

    // Calculate if the pixel is part of the background
    for( int i = index; i < length; i += stride )
    {
        if( cudaImgArr[ i ] < threshold )
        {
            cudaImgArr[ i ] = 0;
        }
    }
}


int* computeMask( int threshold, int imgArr[], int width, int height )
{   
    int length = width * height; // Length of the array
    int blockSize = 256; // Number of threads to use per block
    int numBlocks = (length + blockSize - 1) / blockSize; // Number of blocks
    const int bytes = length * sizeof( int ); // Number of bytes to be allocated
    int *cudaImgArr;

    // Allocate GPU memory send the image argument to device accessible memory
    hipMallocManaged( &cudaImgArr, bytes);
    hipMemcpy( cudaImgArr, imgArr, bytes, hipMemcpyHostToDevice );

    // Perform calculations on GPU
    applyMask<<<numBlocks, blockSize>>>( threshold, length, cudaImgArr );

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Transfer the cuda var back to the host
    hipMemcpy( imgArr, cudaImgArr, bytes, hipMemcpyDeviceToHost );

    // Free the memory
    hipFree( cudaImgArr );

    return imgArr;
}
